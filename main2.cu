#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

#include "./types.h"
#include "./csr_product.cuh"

#include "./utils.h"

int main(int argc, char *argv[]){
  if (argc < 2) {
    printf("./programa blFil blCol \n");
    exit(1);
  }

  srand(0); // Inicializa la semilla aleatoria

  unsigned int blFilN = atoi(argv[1]);
  unsigned int blColN = atoi(argv[2]);

  BlMat A;

  gen_matriz_bloques(&A, blFilN, blColN);

  VALUE *vector = (VALUE*) malloc(A.blColN*8*sizeof(VALUE));

  random_vector(vector, A.nnz);

  VALUE *d_res;
  CUDA_CHK(hipMalloc((void **)&d_res, A.blColN*8*sizeof(VALUE)));

  int *d_blStart;
  CUDA_CHK(hipMalloc((void **)&d_blStart, (A.nBlocks+1)*sizeof(int)));
  CUDA_CHK(hipMemcpy(d_blStart, A.blStart, (A.nBlocks+1)*sizeof(int), hipMemcpyHostToDevice));

  int *d_blColIdx;
  CUDA_CHK(hipMalloc((void **)&d_blColIdx, A.nBlocks*sizeof(int)));
  CUDA_CHK(hipMemcpy(d_blColIdx, A.blColIdx, A.nBlocks*sizeof(int), hipMemcpyHostToDevice));

  unsigned long long *d_blBmp;
  CUDA_CHK(hipMalloc((void **)&d_blBmp, A.nBlocks*sizeof(unsigned long long)));
  CUDA_CHK(hipMemcpy(d_blBmp, A.blBmp, A.nBlocks*sizeof(unsigned long long), hipMemcpyHostToDevice));

  int *d_blRowPtr;
  CUDA_CHK(hipMalloc((void **)&d_blRowPtr, (A.blFilN+1)*sizeof(int)));
  CUDA_CHK(hipMemcpy(d_blRowPtr, A.blRowPtr, (A.blFilN+1)*sizeof(int), hipMemcpyHostToDevice));

  VALUE *d_val;
  CUDA_CHK(hipMalloc((void **)&d_val, A.nnz*sizeof(VALUE)));
  CUDA_CHK(hipMemcpy(d_val, A.val, A.nnz*sizeof(VALUE), hipMemcpyHostToDevice));

  A.blStart = d_blStart;
  A.blColIdx = d_blColIdx;
  A.blBmp = d_blBmp;
  A.blRowPtr = d_blRowPtr;
  A.val = d_val;

  VALUE *d_vector;
  CUDA_CHK(hipMalloc((void **)&d_vector, A.blColN*8*sizeof(VALUE)));
  CUDA_CHK(hipMemcpy(d_vector, vector, A.blColN*8*sizeof(VALUE), hipMemcpyHostToDevice));

	dim3 dimBlock(256);
  // Fast ceil(A_csr.colN/256)
	dim3 dimGrid((A.blFilN + 1 + 256 - 1) / 256);

  // spmv_csr_kernel<<<dimGrid, dimBlock>>>(A_csr, d_vector, d_res);
  bsr_vector_kernel<<<dimGrid, dimBlock>>>(A, d_vector, d_res);

  CUDA_CHK(hipGetLastError());
  CUDA_CHK(hipDeviceSynchronize());
  VALUE *res = (VALUE*) malloc(A.blColN*8*sizeof(VALUE));
  CUDA_CHK(hipMemcpy(res, d_res, A.blColN*8*sizeof(VALUE), hipMemcpyDeviceToHost));

  printf("\n");

  for (int i = 0; i < 5; ++i)
  {
    printf("%.2f\n", res[i]);
  }

  hipFree(d_vector);
  hipFree(d_res);
  hipFree(d_blStart);
  hipFree(d_blColIdx);
  hipFree(d_blBmp);
  hipFree(d_blRowPtr);
  hipFree(d_val);

	free(vector);
  free(res);
	return 0;
}
